#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>

#include "encode_gpu.h"

#define RUN(x) (run(x, __FILE__, __LINE__))

__host__ void run(hipError_t err, const char *file, int line)
{
	if (err) {
		fprintf(stderr, "Error in file %s at line %d:\n%s\n%s\n",
				file, line, hipGetErrorName(err), hipGetErrorString(err));
		EXIT;
	}
}

__device__ void merge(const byte_t *const left_bytes, const natural_t left_size,
					  const byte_t *const right_bytes, const natural_t right_size,
					  byte_t *&output_bytes, natural_t output_size)
{
	output_size = 100;
	output_bytes = MALLOC(output_size);
}

__global__ void kernel(const natural_t input_size)
{
	natural_t temp_size, index;
	byte_t *temp_bytes;

	index = blockDim.x * blockIdx.x + threadIdx.x;

	temp_size = input_size;
	temp_bytes = (byte_t *)(input_size * index);



	// merge see github c#
	//if (index % ? == ?) { ...
}

__host__ void encode_gpu(const byte_t *const input_bytes,
						 const natural_t input_size,
						 byte_t *&output_bytes, natural_t &output_size)
{
	byte_t *input_bytes_d = NULL;
	RUN(hipMalloc((void **)&input_bytes_d, input_size));
	RUN(hipMemcpy(input_bytes_d, input_bytes, input_size,
				   hipMemcpyHostToDevice));

	byte_t *temp_d;
	RUN(hipMalloc((void **)&temp_d, 32 * 2 * input_size));

	kernel<<<1, 32>>>(input_size);
	RUN(hipGetLastError());

	// extract output_size
	output_size = 100;
	output_bytes = MALLOC(output_size);

	RUN(hipMemcpy(output_bytes, temp_d, output_size, hipMemcpyDeviceToHost));

	output_size = 0;
	output_bytes = nullptr;
}
